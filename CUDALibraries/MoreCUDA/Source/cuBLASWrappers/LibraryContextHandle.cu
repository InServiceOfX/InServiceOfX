#include "cuBLASWrappers/LibraryContextHandle.h"

#include <hipblaslt.h>
#include <iostream> // std::cerr
#include <stdexcept>
#include <string>

using std::cerr;
using std::string;

namespace cuBLASWrappers
{

LibraryContextHandle::LibraryContextHandle():
  handle_{}
{
  handle_creation(create_handle());
}

LibraryContextHandle::~LibraryContextHandle()
{
  handle_destruction(destroy_handle());
}

//------------------------------------------------------------------------------
/// https://docs.nvidia.com/cuda/cublas/#cublasltcreate
/// hipblasStatus_t hipblasLtCreate(hipblasLtHandle_t *lighthandle)
//------------------------------------------------------------------------------
hipblasStatus_t LibraryContextHandle::create_handle()
{
  return hipblasLtCreate(&handle_);
}
//------------------------------------------------------------------------------
/// https://docs.nvidia.com/cuda/cublas/#cublasltdestroy
/// hipblasStatus_t hipblasLtDestroy(hipblasLtHandle_t lighthandle)
//------------------------------------------------------------------------------
hipblasStatus_t LibraryContextHandle::destroy_handle()
{
  return hipblasLtDestroy(handle_);
}

// https://docs.nvidia.com/cuda/cublas/#cublasltcreate
bool LibraryContextHandle::handle_creation(hipblasStatus_t status)
{
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    if (status == HIPBLAS_STATUS_NOT_INITIALIZED)
    {
      std::string error_message {
        "cuBLASLt library was not initialized. This usually happens: error in "
        "CUDA Runtime API, or error in hardware setup."};
      
      throw std::runtime_error(error_message);
    }
    else if (status == HIPBLAS_STATUS_ALLOC_FAILED)
    {
      std::string error_message {
        "Resource allocation failed inside cuBLASLt library. This is usually "
        "caused by a hipMalloc() failure.\n\nTo correct: prior to function "
        "call, deallocate previously allocated memory as much as possible."};

      throw std::runtime_error(error_message);
    }
    else if (status == HIPBLAS_STATUS_INVALID_VALUE)
    {
      throw std::runtime_error("lighthandle == NULL");
    }
    else
    {
      throw std::runtime_error("Unknown cuBLASLt library error.");
    }
  }

  return status == HIPBLAS_STATUS_SUCCESS;
}

bool LibraryContextHandle::handle_destruction(hipblasStatus_t status)
{
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    if (status == HIPBLAS_STATUS_NOT_INITIALIZED)
    {
      static constexpr const char* error_message {
        "cuBLASLt library was not initialized."};

      cerr << error_message << '\n';
    }
    else if (status == HIPBLAS_STATUS_INVALID_VALUE)
    {
      cerr << "lighthandle == NULL" << '\n';
    }
    else
    {
      cerr << "Unknown cuBLASLt library error.\n";
    }
  }

  return status == HIPBLAS_STATUS_SUCCESS;
}

} // namespace cuBLASWrappers
