#include "cuBLASWrappers/MatrixMultiplication/cuBLASLtDescriptor.h"

#include <hipblaslt.h>
#include <stdexcept>

namespace cuBLASWrappers
{
namespace MatrixMultiplication
{

cuBLASLtDescriptor::cuBLASLtDescriptor():
  descriptor_{}
{
  create_descriptor(get_compute_parameters<float>());
}

cuBLASLtDescriptor::cuBLASLtDescriptor(
	const ComputeParameters compute_parameters):
  descriptor_{}
{
  create_descriptor(compute_parameters);
}

cuBLASLtDescriptor::~cuBLASLtDescriptor()
{
  destroy_descriptor();
}

bool cuBLASLtDescriptor::create_descriptor(
  const ComputeParameters compute_parameters)
{
	// https://docs.nvidia.com/cuda/cublas/#cublasltmatmuldesccreate
	// hipblasStatus_t hipblasLtMatmulDescCreate(cublashLtMatmulDesc_t *matmulDesc,
  //   hipblasComputeType_t computeType,
	//   hipDataType scaleType)
  const hipblasStatus_t status {
    hipblasLtMatmulDescCreate(
      &descriptor_,
      compute_parameters.compute_precision_mode_,
      compute_parameters.data_type_)};

  if (status != HIPBLAS_STATUS_SUCCESS)
  {
		if (status == HIPBLAS_STATUS_ALLOC_FAILED)
		{
			throw std::runtime_error("Memory could not be allocated.");
		}

    throw std::runtime_error("Failed to create cuBLASLt descriptor");
  }

  return status == HIPBLAS_STATUS_SUCCESS;
}

bool cuBLASLtDescriptor::destroy_descriptor()
{
  return hipblasLtMatmulDescDestroy(descriptor_) == HIPBLAS_STATUS_SUCCESS;
}

} // namespace MatrixMultiplication
} // namespace cuBLASWrappers
