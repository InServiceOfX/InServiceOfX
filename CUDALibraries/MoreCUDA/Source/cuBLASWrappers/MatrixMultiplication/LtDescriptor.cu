#include "cuBLASWrappers/MatrixMultiplication/LtDescriptor.h"

#include <hipblaslt.h>
#include <stdexcept>

namespace cuBLASWrappers
{
namespace MatrixMultiplication
{

LtDescriptor::LtDescriptor():
  descriptor_{}
{
  create_descriptor(get_compute_parameters<float>());
}

LtDescriptor::LtDescriptor(
	const ComputeParameters compute_parameters):
  descriptor_{}
{
  create_descriptor(compute_parameters);
}

LtDescriptor::~LtDescriptor()
{
  destroy_descriptor();
}

bool LtDescriptor::create_descriptor(
  const ComputeParameters compute_parameters)
{
	// https://docs.nvidia.com/cuda/cublas/#cublasltmatmuldesccreate
	// hipblasStatus_t hipblasLtMatmulDescCreate(cublashLtMatmulDesc_t *matmulDesc,
  //   hipblasComputeType_t computeType,
	//   hipDataType scaleType)
  const hipblasStatus_t status {
    hipblasLtMatmulDescCreate(
      &descriptor_,
      compute_parameters.compute_precision_mode_,
      compute_parameters.data_type_)};

  if (status != HIPBLAS_STATUS_SUCCESS)
  {
		if (status == HIPBLAS_STATUS_ALLOC_FAILED)
		{
			throw std::runtime_error("Memory could not be allocated.");
		}

    throw std::runtime_error("Failed to create cuBLASLt descriptor");
  }

  return status == HIPBLAS_STATUS_SUCCESS;
}

bool LtDescriptor::destroy_descriptor()
{
  return hipblasLtMatmulDescDestroy(descriptor_) == HIPBLAS_STATUS_SUCCESS;
}

} // namespace MatrixMultiplication
} // namespace cuBLASWrappers
